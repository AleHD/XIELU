#include "hip/hip_runtime.h"
#include "utils.cuh"
#include "xielu_impl.hpp"

#include <iostream>

#include <ATen/cuda/Atomic.cuh>
#include <c10/cuda/CUDAGuard.h>
#include <c10/cuda/CUDAStream.h>
#include <hip/hip_bf16.h>
#include <hip/hip_fp16.h>
#include <math.h>
#include <torch/script.h>

using namespace std;
using namespace torch::indexing;
using namespace torch::autograd;
using namespace c10;

using torch::Tensor;
using torch::TensorOptions;

#define NWARPS 8
#define WARP_SIZE 32

#define CHECK_RESULT(result)                                                   \
  if (result != hipSuccess) {                                                 \
    throw runtime_error(string("Encountered error ") +                         \
                        hipGetErrorName(result) + " at " + __FILE__ + ": " +  \
                        to_string(__LINE__));                                  \
  }

static int getMaxBlocks() {
  // Get an upper limit on how many thread blocks we try to launch based on the
  // size of the GPU.
  int device, numMultiprocessors;
  CHECK_RESULT(hipGetDevice(&device));
  CHECK_RESULT(hipDeviceGetAttribute(&numMultiprocessors,
                                      hipDeviceAttributeMultiprocessorCount, device));
  return numMultiprocessors * 4;
}

/* specialized structure for vectorised loads with half, bfloat16 types */
template <typename T> struct vec4 { T x, y, z, w; };

using half4 = vec4<c10::Half>;
using bfloat4 = vec4<c10::BFloat16>;

template <typename T> struct softplus {
  static __device__ T f(T x) {
    return x > T(20.0) ? x : (x < T(-20.0) ? T(0.0) : log1p(exp(x)));
  }

  static __device__ T df(T x) {
    return x > T(20.0) ? T(1.0)
                       : (x < T(-20.0) ? T(0.0) : T(1.0) / (T(1.0) + exp(-x)));
  }
};

template <> struct softplus<c10::Half> {
  static __device__ c10::Half f(c10::Half x) {
    return static_cast<c10::Half>(softplus<float>::f(static_cast<float>(x)));
  }
  static __device__ c10::Half df(c10::Half x) {
    return static_cast<c10::Half>(softplus<float>::df(static_cast<float>(x)));
  }
};

template <> struct softplus<c10::BFloat16> {
  static __device__ c10::BFloat16 f(c10::BFloat16 x) {
    return static_cast<c10::BFloat16>(
        softplus<float>::f(static_cast<float>(x)));
  }
  static __device__ c10::BFloat16 df(c10::BFloat16 x) {
    return static_cast<c10::BFloat16>(
        softplus<float>::df(static_cast<float>(x)));
  }
};

// Generic overload: cast to float
template <typename T> __device__ __forceinline__ float to_float_if_needed(T x) {
  return static_cast<float>(x);
}

// Overload for float: return as-is
__device__ __forceinline__ float to_float_if_needed(float x) { return x; }

// Overload for double: return as-is
__device__ __forceinline__ double to_float_if_needed(double x) { return x; }

template <typename scalar_t>
__device__ __forceinline__ scalar_t compute_expm1(const scalar_t x) {
  float x_f = to_float_if_needed(x);
  return static_cast<scalar_t>(expf(x_f) - 1.0f);
}

template <typename scalar_t>
__device__ __forceinline__ scalar_t compute(scalar_t v,
                                            const scalar_t s_alpha_p,
                                            const scalar_t s_alpha_n,
                                            const scalar_t beta,
                                            const scalar_t eps) {

  return to_float_if_needed(v) > scalar_t(0.0)
             ? v * (s_alpha_p * v + beta)
             : s_alpha_n * compute_expm1<scalar_t>(min(v, eps)) -
                   (beta + s_alpha_n) * v;
}

template <typename scalar_t, typename vector_t>
__global__ void vectorised_xielu_forward_impl(
    const scalar_t *__restrict__ x, const int total_elements,
    const Accessor<scalar_t, 1> alpha_p, const Accessor<scalar_t, 1> alpha_n,
    const scalar_t beta, const scalar_t eps, scalar_t *__restrict__ output) {
  using sp = softplus<scalar_t>;
  const scalar_t s_alpha_p = sp::f(alpha_p[0]);
  const scalar_t s_alpha_n = sp::f(alpha_n[0]);

  for (int vec_idx = blockIdx.x * blockDim.x + threadIdx.x;
       vec_idx < total_elements / 4; vec_idx += blockDim.x * gridDim.x) {

    int base_idx = vec_idx * 4;

    vector_t x_v = *reinterpret_cast<const vector_t *>(&x[base_idx]);
    vector_t out;

    out.x = compute(x_v.x, s_alpha_p, s_alpha_n, beta, eps);
    out.y = compute(x_v.y, s_alpha_p, s_alpha_n, beta, eps);
    out.z = compute(x_v.z, s_alpha_p, s_alpha_n, beta, eps);
    out.w = compute(x_v.w, s_alpha_p, s_alpha_n, beta, eps);

    *reinterpret_cast<vector_t *>(&output[base_idx]) = out;
  }
}

template <typename scalar_t>
__global__ void
xielu_forward_impl(const scalar_t *__restrict__ x, const int total_elements,
                   const Accessor<scalar_t, 1> alpha_p,
                   const Accessor<scalar_t, 1> alpha_n, const scalar_t beta,
                   const scalar_t eps, scalar_t *__restrict__ output) {
  using sp = softplus<scalar_t>;
  const scalar_t s_alpha_p = sp::f(alpha_p[0]);
  const scalar_t s_alpha_n = sp::f(alpha_n[0]);

  for (int i = blockIdx.x * blockDim.x + threadIdx.x; i < total_elements;
       i += blockDim.x * gridDim.x) {
    scalar_t x_v = x[i];
    scalar_t out = compute(x_v, s_alpha_p, s_alpha_n, beta, eps);
    output[i] = out;
  }
}

torch::Tensor XIELUAutograd::forward(AutogradContext *ctx, Tensor x,
                                     Tensor alpha_p, Tensor alpha_n,
                                     double beta, double eps,
                                     bool with_vector_loads) {

  PUSH_RANGE("XIELU_FWD", 0)

  TORCH_CHECK(x.is_cuda(), "Input tensor x must be on the CUDA device.");
  TORCH_CHECK(alpha_p.is_cuda(),
              "Input tensor alpha_p must be on the CUDA device.");
  TORCH_CHECK(alpha_n.is_cuda(),
              "Input tensor alpha_n must be on the CUDA device.");
  TORCH_CHECK(alpha_p.dim() == 1 && alpha_p.numel() == 1,
              "alpha_p must be a 1-D tensor with one element.");
  TORCH_CHECK(alpha_n.dim() == 1 && alpha_n.numel() == 1,
              "alpha_n must be a 1-D tensor with one element.");
  TORCH_CHECK(x.dtype() == alpha_p.dtype(), "Data type of x (", x.dtype(),
              ") must match data type of alpha_p (", alpha_p.dtype(), ").");
  TORCH_CHECK(x.dtype() == alpha_n.dtype(), "Data type of x (", x.dtype(),
              ") must match data type of alpha_n (", alpha_n.dtype(), ").");

  const int batch_size = x.size(0);
  const int seq_len = x.size(1);
  const int hidden_dim = x.size(2);
  const int nelements = batch_size * seq_len * hidden_dim;

  TORCH_CHECK(hidden_dim % 4 == 0, "hidden_dim must be a multiple of 4");

  const int blockSize = NWARPS * WARP_SIZE;
  const int elements_per_thread = with_vector_loads ? 4 : 1;
  const int adjusted_elements = nelements / elements_per_thread;
  const int numBlocks = max(
      1, min(getMaxBlocks(), (adjusted_elements + blockSize - 1) / blockSize));

  TensorOptions options = x.options();
  Tensor output = torch::empty_like(x);

  const auto stream = c10::cuda::getCurrentCUDAStream(x.get_device());
  const c10::cuda::CUDAStreamGuard guard(stream);

  AT_DISPATCH_FLOATING_TYPES_AND2(
      at::ScalarType::Half, at::ScalarType::BFloat16, x.scalar_type(),
      "forward", ([&] {
        using vector_t = typename std::conditional<
            std::is_same<scalar_t, float>::value, float4,
            typename std::conditional<
                std::is_same<scalar_t, double>::value, double4,
                typename std::conditional<
                    std::is_same<scalar_t, c10::Half>::value, half4,
                    typename std::conditional<
                        std::is_same<scalar_t, c10::BFloat16>::value, bfloat4,
                        void>::type>::type>::type>::type;

        static_assert(!std::is_same<vector_t, void>::value, "Unsupported type");

        if (with_vector_loads) {
          vectorised_xielu_forward_impl<scalar_t, vector_t>
              <<<numBlocks, blockSize, 0, stream>>>(
                  x.data_ptr<scalar_t>(), nelements,
                  get_accessor<scalar_t, 1>(alpha_p),
                  get_accessor<scalar_t, 1>(alpha_n), (scalar_t)beta,
                  (scalar_t)eps, output.data_ptr<scalar_t>());
        } else {

          xielu_forward_impl<scalar_t><<<numBlocks, blockSize, 0, stream>>>(
              x.data_ptr<scalar_t>(), nelements,
              get_accessor<scalar_t, 1>(alpha_p),
              get_accessor<scalar_t, 1>(alpha_n), (scalar_t)beta, (scalar_t)eps,
              output.data_ptr<scalar_t>());
        }
      }));

  ctx->save_for_backward({x, alpha_p, alpha_n});
  ctx->saved_data["eps"] = eps;
  ctx->saved_data["beta"] = beta;
  ctx->saved_data["with_vector_loads"] = with_vector_loads;

  POP_RANGE

  return output;
}

template <typename scalar_t>
__device__ __forceinline__ scalar_t compute_dx(scalar_t v, scalar_t dout,
                                               scalar_t s_alpha_p,
                                               scalar_t s_alpha_n,
                                               scalar_t beta, scalar_t eps) {
  return to_float_if_needed(v) > scalar_t(0.0)
             ? dout * (2 * s_alpha_p * v + beta)
             : dout * (s_alpha_n * compute_expm1<scalar_t>(min(v, eps)) + beta);
}

template <typename scalar_t>
__device__ __forceinline__ scalar_t compute_dp(scalar_t v, scalar_t dout,
                                               scalar_t ds_alpha_p) {
  return to_float_if_needed(v) > scalar_t(0.0) ? dout * ds_alpha_p * v * v
                                               : scalar_t(0.0);
}

template <typename scalar_t>
__device__ __forceinline__ scalar_t compute_dn(scalar_t v, scalar_t dout,
                                               scalar_t ds_alpha_n,
                                               scalar_t eps) {
  return to_float_if_needed(v) <= scalar_t(0.0)
             ? dout * ds_alpha_n * (compute_expm1<scalar_t>(min(v, eps)) - v)
             : scalar_t(0.0);
}

template <typename scalar_t, typename reduction_type, typename vector_t>
__global__ void vectorised_xielu_backward_impl(
    const scalar_t *__restrict__ x, const int total_elements,
    const Accessor<scalar_t, 1> alpha_p, const Accessor<scalar_t, 1> alpha_n,
    const scalar_t *__restrict__ grad_outputs, const scalar_t beta,
    const scalar_t eps, scalar_t *__restrict__ dx,
    Accessor<reduction_type, 1> dalpha_p,
    Accessor<reduction_type, 1> dalpha_n) {

  using sp = softplus<scalar_t>;
  const scalar_t _alpha_p = alpha_p[0];
  const scalar_t _alpha_n = alpha_n[0];

  const scalar_t s_alpha_p = sp::f(_alpha_p);
  const scalar_t s_alpha_n = beta + sp::f(_alpha_n);
  const scalar_t ds_alpha_p = sp::df(_alpha_p);
  const scalar_t ds_alpha_n = sp::df(_alpha_n);

  reduction_type thread_dalpha_p = reduction_type(0.0);
  reduction_type thread_dalpha_n = reduction_type(0.0);

  for (int vec_idx = blockIdx.x * blockDim.x + threadIdx.x;
       vec_idx < total_elements / 4; vec_idx += blockDim.x * gridDim.x) {

    int base_idx = vec_idx * 4;

    vector_t x_v = *reinterpret_cast<const vector_t *>(&x[base_idx]);

    vector_t grad_output_v =
        *reinterpret_cast<const vector_t *>(&grad_outputs[base_idx]);
    vector_t dx_v;
    vector_t dalpha_p_v, dalpha_n_v;

    dx_v.x =
        compute_dx(x_v.x, grad_output_v.x, s_alpha_p, s_alpha_n, beta, eps);
    dx_v.y =
        compute_dx(x_v.y, grad_output_v.y, s_alpha_p, s_alpha_n, beta, eps);
    dx_v.z =
        compute_dx(x_v.z, grad_output_v.z, s_alpha_p, s_alpha_n, beta, eps);
    dx_v.w =
        compute_dx(x_v.w, grad_output_v.w, s_alpha_p, s_alpha_n, beta, eps);

    dalpha_p_v.x = compute_dp(x_v.x, grad_output_v.x, ds_alpha_p);
    dalpha_p_v.y = compute_dp(x_v.y, grad_output_v.y, ds_alpha_p);
    dalpha_p_v.z = compute_dp(x_v.z, grad_output_v.z, ds_alpha_p);
    dalpha_p_v.w = compute_dp(x_v.w, grad_output_v.w, ds_alpha_p);

    dalpha_n_v.x = compute_dn(x_v.x, grad_output_v.x, ds_alpha_n, eps);
    dalpha_n_v.y = compute_dn(x_v.y, grad_output_v.y, ds_alpha_n, eps);
    dalpha_n_v.z = compute_dn(x_v.z, grad_output_v.z, ds_alpha_n, eps);
    dalpha_n_v.w = compute_dn(x_v.w, grad_output_v.w, ds_alpha_n, eps);

    *reinterpret_cast<vector_t *>(&dx[base_idx]) = dx_v;

    thread_dalpha_p +=
        dalpha_p_v.x + dalpha_p_v.y + dalpha_p_v.z + dalpha_p_v.w;
    thread_dalpha_n +=
        dalpha_n_v.x + dalpha_n_v.y + dalpha_n_v.z + dalpha_n_v.w;
  }

  __syncthreads();

  // reduce thread-local contributions into thread % 32 = 0
  for (int offset = WARP_SIZE / 2; offset > 0; offset /= 2) {
    thread_dalpha_p += __shfl_down_sync(0xffffffff, thread_dalpha_p, offset);
    thread_dalpha_n += __shfl_down_sync(0xffffffff, thread_dalpha_n, offset);
  }

  // write each warp's contributions to grad to gmem
  if (threadIdx.x % WARP_SIZE == 0) {
    gpuAtomicAdd(&dalpha_p[0], thread_dalpha_p);
    gpuAtomicAdd(&dalpha_n[0], thread_dalpha_n);
  }
}

template <typename scalar_t, typename reduction_type>
__global__ void xielu_backward_impl(const scalar_t *__restrict__ x,
                                    const int total_elements,
                                    const Accessor<scalar_t, 1> alpha_p,
                                    const Accessor<scalar_t, 1> alpha_n,
                                    const scalar_t *__restrict__ grad_outputs,
                                    const scalar_t beta, const scalar_t eps,
                                    scalar_t *__restrict__ dx,
                                    Accessor<reduction_type, 1> dalpha_p,
                                    Accessor<reduction_type, 1> dalpha_n) {

  using sp = softplus<scalar_t>;
  const scalar_t _alpha_p = alpha_p[0];
  const scalar_t _alpha_n = alpha_n[0];

  const scalar_t s_alpha_p = sp::f(_alpha_p);
  const scalar_t s_alpha_n = beta + sp::f(_alpha_n);
  const scalar_t ds_alpha_p = sp::df(_alpha_p);
  const scalar_t ds_alpha_n = sp::df(_alpha_n);

  reduction_type thread_dalpha_p = reduction_type(0.0);
  reduction_type thread_dalpha_n = reduction_type(0.0);

  for (int i = blockIdx.x * blockDim.x + threadIdx.x; i < total_elements;
       i += blockDim.x * gridDim.x) {

    scalar_t x_v = x[i];
    scalar_t grad_output_v = grad_outputs[i];

    scalar_t dx_v =
        compute_dx(x_v, grad_output_v, s_alpha_p, s_alpha_n, beta, eps);
    scalar_t dalpha_p_v = compute_dp(x_v, grad_output_v, ds_alpha_p);
    scalar_t dalpha_n_v = compute_dn(x_v, grad_output_v, ds_alpha_n, eps);

    dx[i] = dx_v;

    thread_dalpha_p += dalpha_p_v;
    thread_dalpha_n += dalpha_n_v;
  }

  __syncthreads();

  // reduce thread-local contributions into thread % 32 = 0
  for (int offset = WARP_SIZE / 2; offset > 0; offset /= 2) {
    thread_dalpha_p += __shfl_down_sync(0xffffffff, thread_dalpha_p, offset);
    thread_dalpha_n += __shfl_down_sync(0xffffffff, thread_dalpha_n, offset);
  }

  // write each warp's contributions to grad to gmem
  if (threadIdx.x % WARP_SIZE == 0) {
    gpuAtomicAdd(&dalpha_p[0], thread_dalpha_p);
    gpuAtomicAdd(&dalpha_n[0], thread_dalpha_n);
  }
}

variable_list XIELUAutograd::backward(AutogradContext *ctx,
                                      variable_list grad_outputs) {

  PUSH_RANGE("XIELU_BWD", 1)

  auto saved = ctx->get_saved_variables();
  Tensor x = saved[0];
  Tensor alpha_p = saved[1];
  Tensor alpha_n = saved[2];
  const double eps = ctx->saved_data["eps"].toDouble();
  const double beta = ctx->saved_data["beta"].toDouble();
  const bool with_vector_loads = ctx->saved_data["with_vector_loads"].toBool();

  TORCH_CHECK(x.is_cuda(), "Input tensor x must be on the CUDA device.");
  TORCH_CHECK(alpha_p.is_cuda(),
              "Input tensor alpha_p must be on the CUDA device.");
  TORCH_CHECK(alpha_n.is_cuda(),
              "Input tensor alpha_n must be on the CUDA device.");
  TORCH_CHECK(alpha_p.dim() == 1 && alpha_p.numel() == 1,
              "alpha_p must be a 1-D tensor with one element.");
  TORCH_CHECK(alpha_n.dim() == 1 && alpha_n.numel() == 1,
              "alpha_n must be a 1-D tensor with one element.");
  TORCH_CHECK(x.dtype() == alpha_p.dtype(), "Data type of x (", x.dtype(),
              ") must match data type of alpha_p (", alpha_p.dtype(), ").");
  TORCH_CHECK(x.dtype() == alpha_n.dtype(), "Data type of x (", x.dtype(),
              ") must match data type of alpha_n (", alpha_n.dtype(), ").");

  const int nbatch = x.size(0);
  const int seq_len = x.size(1);
  const int hidden_dim = x.size(2);
  const int nelements = nbatch * seq_len * hidden_dim;

  TORCH_CHECK(hidden_dim % 4 == 0, "hidden_dim must be a multiple of 4");

  const int blockSize = NWARPS * WARP_SIZE;
  const int elements_per_thread = with_vector_loads ? 4 : 1;
  const int adjusted_elements = nelements / elements_per_thread;
  const int numBlocks = max(
      1, min(getMaxBlocks(), (adjusted_elements + blockSize - 1) / blockSize));

  TensorOptions options = x.options();

  Tensor dx = torch::empty_like(x);
  // each block stores a contribution to dalpha_p, dalpha_n
  // do reductions to dalpha_p, dalpha_n in higher precision to avoid
  // numerical errors
  Tensor dalpha_p;
  Tensor dalpha_n;

  if (x.scalar_type() == at::ScalarType::Half ||
      x.scalar_type() == at::ScalarType::BFloat16) {
    dalpha_p = torch::zeros({1}, options.dtype(torch::kFloat32));
    dalpha_n = torch::zeros({1}, options.dtype(torch::kFloat32));
  } else {
    dalpha_p = torch::zeros({1}, options);
    dalpha_n = torch::zeros({1}, options);
  }

  if (!grad_outputs[0].is_contiguous())
    grad_outputs[0] = grad_outputs[0].contiguous();

  const auto stream = c10::cuda::getCurrentCUDAStream(x.get_device());
  const c10::cuda::CUDAStreamGuard guard(stream);

  AT_DISPATCH_FLOATING_TYPES_AND2(
      at::ScalarType::Half, at::ScalarType::BFloat16, x.scalar_type(),
      "backward", ([&] {
        using vector_t = typename std::conditional<
            std::is_same<scalar_t, float>::value, float4,
            typename std::conditional<
                std::is_same<scalar_t, double>::value, double4,
                typename std::conditional<
                    std::is_same<scalar_t, c10::Half>::value, half4,
                    typename std::conditional<
                        std::is_same<scalar_t, c10::BFloat16>::value, bfloat4,
                        void>::type>::type>::type>::type;

        static_assert(!std::is_same<vector_t, void>::value, "Unsupported type");

        using reduction_t = typename std::conditional<
            std::is_same<scalar_t, c10::Half>::value ||
                std::is_same<scalar_t, c10::BFloat16>::value,
            float, scalar_t>::type;

        if (with_vector_loads) {
          vectorised_xielu_backward_impl<scalar_t, reduction_t, vector_t>
              <<<numBlocks, blockSize, 0, stream>>>(
                  x.data_ptr<scalar_t>(), nelements,
                  get_accessor<scalar_t, 1>(alpha_p),
                  get_accessor<scalar_t, 1>(alpha_n),
                  grad_outputs[0].data_ptr<scalar_t>(), (scalar_t)beta,
                  (scalar_t)eps, dx.data_ptr<scalar_t>(),
                  get_accessor<reduction_t, 1>(dalpha_p),
                  get_accessor<reduction_t, 1>(dalpha_n));
        } else {
          xielu_backward_impl<scalar_t, reduction_t>
              <<<numBlocks, blockSize, 0, stream>>>(
                  x.data_ptr<scalar_t>(), nelements,
                  get_accessor<scalar_t, 1>(alpha_p),
                  get_accessor<scalar_t, 1>(alpha_n),
                  grad_outputs[0].data_ptr<scalar_t>(), (scalar_t)beta,
                  (scalar_t)eps, dx.data_ptr<scalar_t>(),
                  get_accessor<reduction_t, 1>(dalpha_p),
                  get_accessor<reduction_t, 1>(dalpha_n));
        }
      }));

  torch::Tensor undef;

  POP_RANGE

  return {dx,   dalpha_p.to(x.dtype()), dalpha_n.to(x.dtype()), undef, undef,
          undef};
}
